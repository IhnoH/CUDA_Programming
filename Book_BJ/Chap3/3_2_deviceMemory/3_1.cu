#include "hip/hip_runtime.h"

#include <stdio.h>

void checkDeviceMemory(void)
{
	size_t free, total;
	hipMemGetInfo(&free, &total); // ���� ��밡���� device memory ũ���, �� device �޸� ũ�⸦ ������ �Լ�
	printf("Device memory (free/total) = %lld/%lld bytes\n"
		, free, total);
}

int main(void)
{
	int* dDataPtr;
	hipError_t errorCode ;

	checkDeviceMemory();
	errorCode = hipMalloc(&dDataPtr, sizeof(int) * 10240); // device memory �Ҵ�
	printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 10240); // device memory �ʱ�ȭ
	printf("hipMemset - %s\n", hipGetErrorName(errorCode));

	errorCode = hipFree(dDataPtr);
	printf("hipFree - %s\n", hipGetErrorName(errorCode)); // device memory ����
	checkDeviceMemory();
}