#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "DS_timer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define ARRAY_SIZE (64*1024*1024)
#define B_SIZE (1024)

#define NUM_STREAMS 1

__global__ void myKernel(int* _in, int* _out)
{
	int tID = blockDim.x * blockIdx.x + threadIdx.x;

	int temp = 0;
	for (int i = 0; i < 250; i++) {
		temp = (temp + _in[tID] * 5) % 10;
	}
	_out[tID] = temp;

}

void main(void)
{
	DS_timer timer(1);
	timer.setTimerName(0, "Total");

	int* in = NULL, * out = NULL, * out2 = NULL;

	hipHostMalloc(&in, sizeof(int) * ARRAY_SIZE);
	memset(in, 0, sizeof(int) * ARRAY_SIZE);

	hipHostMalloc(&out, sizeof(int) * ARRAY_SIZE);
	memset(out, 0, sizeof(int) * ARRAY_SIZE);

	hipHostMalloc(&out2, sizeof(int) * ARRAY_SIZE);
	memset(out2, 0, sizeof(int) * ARRAY_SIZE);

	int* dIn, * dOut;
	hipMalloc(&dIn, sizeof(int) * ARRAY_SIZE);
	hipMalloc(&dOut, sizeof(int) * ARRAY_SIZE);

	for (int i = 0; i < ARRAY_SIZE; i++)
		in[i] = rand() % 10;

	// Multiple stream version
	hipStream_t stream[NUM_STREAMS];
	hipEvent_t start[NUM_STREAMS], end[NUM_STREAMS];

	for (int i = 0; i < NUM_STREAMS; i++) {
		hipStreamCreate(&stream[i]);
		hipEventCreate(&start[i]); hipEventCreate(&end[i]);
	}

	int chunkSize = ARRAY_SIZE / NUM_STREAMS;

	timer.onTimer(0);
	for (int i = 0; i < NUM_STREAMS; i++)
	{
		int offset = chunkSize * i;
		hipEventRecord(start[i], stream[i]);

		hipMemcpyAsync(dIn + offset, in + offset, sizeof(int) * chunkSize, hipMemcpyHostToDevice, stream[i]);
		myKernel <<<chunkSize/B_SIZE, B_SIZE, 0, stream[i] >>> (dIn + offset, dOut + offset);
		hipMemcpyAsync(out2 + offset, dOut + offset, sizeof(int) * chunkSize, hipMemcpyDeviceToHost, stream[i]);

		hipEventRecord(end[i], stream[i]);
	}

	hipDeviceSynchronize();
	timer.offTimer(0);
	timer.printTimer();

	for (int i = 0; i < NUM_STREAMS; i++) {
		float time = 0;
		hipEventElapsedTime(&time, start[i], end[i]);
		printf("Stream[%d] : %f ms\n", i, time);
	}

	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (out[i] != out2[i])
			printf("!");
	}

	for (int i = 0; i < NUM_STREAMS; i++) {
		hipStreamDestroy(stream[i]);
		hipEventDestroy(start[i]);
		hipEventDestroy(end[i]);
	}

	hipFree(dIn);
	hipFree(dOut);

	hipHostFree(in);
	hipHostFree(out);
	hipHostFree(out2);
}